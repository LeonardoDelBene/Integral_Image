#include <iostream>
#include <random>
#include <hip/hip_runtime.h>
#include <chrono>
#include <vector>
#include <algorithm>

using namespace std;
using namespace chrono;

#define N 16384 // Numero di righe della matrice
#define M 16384 // Numero di colonne della matrice
#define BLOCK_SIZE 1024

__global__ void scan_kernel(int *d_in, int *d_out, int *block_sums, int m) {
    extern __shared__ int temp[];
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;


    if (idx < m) {
        temp[tid] = d_in[idx];
    } else {
        temp[tid] = 0;
    }
    __syncthreads();

    // Up-sweep
    for (int offset = 1; offset < blockDim.x; offset *= 2) {
        int index = (tid + 1) * offset * 2 - 1;
        __syncthreads();
        if (index < blockDim.x) {
            temp[index] += temp[index - offset];
        }
    }

    // Down-sweep
    for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
        int index = (tid + 1) * offset * 2 - 1;
        __syncthreads();
        if (index + offset < blockDim.x) {
            temp[index + offset] += temp[index];
        }
    }
    __syncthreads();


    if (tid == blockDim.x - 1) {
        block_sums[blockIdx.x] = temp[tid];
    }


    if (idx < m) {
        d_out[idx] = temp[tid];
    }
}

__global__ void scan_block_sums_kernel(int *d_block_sums, int num_blocks) {
    extern __shared__ int temp[];
    int tid = threadIdx.x;

    if (tid < num_blocks) {
        temp[tid] = d_block_sums[tid];
    }
    __syncthreads();


    for (int offset = 1; offset < num_blocks; offset *= 2) {
        int val = 0;
        if (tid >= offset) {
            val = temp[tid - offset];
        }
        __syncthreads();
        if (tid >= offset) {
            temp[tid] += val;
        }
        __syncthreads();
    }

    if (tid < num_blocks) {
        d_block_sums[tid] = temp[tid];
    }
}

__global__ void add_block_sums(int *d_out, int *block_sums_scanned, int m) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (blockIdx.x > 0 && idx < m) {
        d_out[idx] += block_sums_scanned[blockIdx.x - 1];
    }
}


__global__ void transpose_kernel(int* d_in, int* d_out, int n, int m) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < m) {
        d_out[col * n + row] = d_in[row * m + col];
    }
}

void scan_matrix(int* h_mat, int n, int m) {
    int *d_mat, *d_out, *d_block_sums, *d_block_sums_scanned;
    int blockSize = BLOCK_SIZE;
    int numBlocks = (m + blockSize - 1) / blockSize;


    hipMalloc((void**)&d_mat, n * m * sizeof(int));
    hipMalloc((void**)&d_out, n * m * sizeof(int));
    hipMalloc((void**)&d_block_sums, numBlocks * sizeof(int));
    hipMalloc((void**)&d_block_sums_scanned, numBlocks * sizeof(int));

    hipMemcpy(d_mat, h_mat, n * m * sizeof(int), hipMemcpyHostToDevice);

    for (int i = 0; i < n; i++) {
        int row_offset = i * m;

        // Esegui lo scan della riga
        scan_kernel<<<numBlocks, blockSize, blockSize * sizeof(int)>>>(d_mat + row_offset, d_out + row_offset, d_block_sums, m);
        hipDeviceSynchronize();



        // Esegui lo scan sui block sums
        scan_block_sums_kernel<<<1, blockSize, numBlocks * sizeof(int)>>>(d_block_sums, numBlocks);
        hipDeviceSynchronize();

        // Aggiungi i prefix sum dei blocchi alla riga
        add_block_sums<<<numBlocks, blockSize>>>(d_out + row_offset, d_block_sums, m);
        hipDeviceSynchronize();
    }


    hipMemcpy(h_mat, d_out, n * m * sizeof(int), hipMemcpyDeviceToHost);


    hipFree(d_mat);
    hipFree(d_out);
    hipFree(d_block_sums);
    hipFree(d_block_sums_scanned);

}




void transpose_matrix(int* h_mat, int* h_transposed, int n, int m) {
    int *d_mat, *d_transposed;

    hipMalloc((void**)&d_mat, n * m * sizeof(int));
    hipMalloc((void**)&d_transposed, m * n * sizeof(int));


    hipMemcpy(d_mat, h_mat, n * m * sizeof(int), hipMemcpyHostToDevice);


    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((m + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);
    transpose_kernel<<<gridDim, blockDim>>>(d_mat, d_transposed, n, m);


    hipMemcpy(h_transposed, d_transposed, m * n * sizeof(int), hipMemcpyDeviceToHost);


    hipFree(d_mat);
    hipFree(d_transposed);
}


int main() {
    int *h_mat = new int[N * M];
    int *transposed_mat = new int[M * N]; // Matrice trasposta

    // Creazione del generatore di numeri casuali
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis(0, 9); // Numeri casuali tra 0 e 9

    // Inizializza la matrice con numeri casuali
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            h_mat[i * M + j] = dis(gen);
        }
    }


    /*std::cout << "Matrice iniziale:" << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            std::cout << h_mat[i * M + j] << " ";
        }
        std::cout << "\n";
    }*/

    auto start = high_resolution_clock::now();

    scan_matrix(h_mat, N, M);
    hipDeviceSynchronize();

    transpose_matrix(h_mat, transposed_mat, N, M);
    hipDeviceSynchronize();

    scan_matrix(transposed_mat, M, N);
    hipDeviceSynchronize();

    transpose_matrix(transposed_mat, h_mat, M, N);
    hipDeviceSynchronize();

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<milliseconds>(stop - start);

    /*std::cout << "\nMatrice finale:" << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            std::cout << h_mat[i * M + j] << " ";
        }
        std::cout << "\n";
    }*/

    std::cout << "\nTempo di esecuzione: " << duration.count() << " ms" << std::endl;

    delete[] h_mat;
    delete[] transposed_mat;
}
